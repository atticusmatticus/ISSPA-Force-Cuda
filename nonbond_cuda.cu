#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nonbond_cuda.h"

#define nDim 3

// CUDA Kernels

__global__ void nonbond_kernel(float *xyz, float *f, float *charges, float *lj_A, float *lj_B, int *ityp, int nAtoms, float rCut2, float lbox, int *NN, int *numNN, int numNNmax, int *nbparm, int nTypes) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int t = threadIdx.x;
	extern __shared__ float xyz_s[];
	int atom1;
	int atom2;
	int it, jt;    // atom type of atom of interest
	float dist2;	
	int i, k;
	int N;
	int start;
	float r[3];
	float r6;
	float fc;
	float flj;
	float hbox;
	int nlj;
	int chunk;


	// copy positions from global memory to shared memory for each block
	chunk = (int) ( (nAtoms*nDim+blockDim.x-1)/blockDim.x);
	for (i=t*chunk;i<(t+1)*chunk;i++) {
		xyz_s[i] = xyz[i];
	}
	__syncthreads();
	// move on
	if (index < nAtoms)
	{
		hbox = lbox/2.0;
		atom1 = index;
		// start position in neighbor list:
		start = atom1*numNNmax;
		// number of atoms in neighbor list:
		N = __ldg(numNN+atom1);
		for (i=0;i<N;i++) {
			atom2 = __ldg(NN+start+i);
			if (atom2 != atom1) {
				// get interaction type
				it = __ldg(ityp+atom1);
				jt = __ldg(ityp+atom2);
				nlj = nTypes*(it-1)+jt-1;
				nlj = __ldg(nbparm+nlj);
				dist2 = 0.0f;
				for (k=0;k<nDim;k++) {
					//r[k] = __ldg(xyz+atom1*nDim+k) - __ldg(xyz+atom2*nDim+k);
					r[k] = xyz_s[atom1*nDim+k] - xyz_s[atom2*nDim+k];
					if (r[k] > hbox) {
//						r[k] -= (int)(temp/lbox+0.5) * lbox;
						r[k] -= lbox;
					} else if (r[k] < -hbox) {
//						r[k] += (int)(-temp/lbox+0.5) * lbox;
						r[k] += lbox;
					}
					dist2 += r[k]*r[k];
				}
				if (dist2 < rCut2) {
					// LJ force
					r6 = powf(dist2,-3.0);
					flj = r6 * (12.0 * __ldg(lj_A+nlj) * r6 - 6.0 * __ldg(lj_B+nlj)) / dist2;
					fc = __ldg(charges+atom1)*__ldg(charges+atom2)/dist2/sqrtf(dist2);
					f[atom1*nDim] += (flj+fc)*r[0];
					f[atom1*nDim+1] += (flj+fc)*r[1];
					f[atom1*nDim+2] += (flj+fc)*r[2];
				}
			}
		}

	}
}

/* C wrappers for kernels */

extern "C" void nonbond_cuda(float *xyz_d, float *f_d, float *charges_d, float *lj_A_d, float *lj_B_d, int *ityp_d, int nAtoms, float rCut2, float lbox, int *NN_d, int *numNN_d, int numNNmax, int *nbparm_d, int nTypes) {
	int blockSize;      // The launch configurator returned block size 
    	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    	int gridSize;       // The actual grid size needed, based on input size 

	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, nonbond_kernel, 0, nAtoms); 

    	// Round up according to array size 
    	gridSize = (nAtoms + blockSize - 1) / blockSize; 

	// run nonbond cuda kernel
	nonbond_kernel<<<gridSize, blockSize, nAtoms*nDim*sizeof(float)>>>(xyz_d, f_d, charges_d, lj_A_d, lj_B_d, ityp_d, nAtoms, rCut2, lbox, NN_d, numNN_d, numNNmax, nbparm_d, nTypes);

}

