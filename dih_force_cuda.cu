#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "dih_force_cuda.h"
#include "constants.h"

//Fast integer multiplication
#define MUL(a, b) __umul24(a, b)

// CUDA Kernels

__global__ void dih_force_kernel(float *xyz, float *f, int nAtoms, float lbox, int *dihAtoms, float *dihKs, float *dihNs, float *dihPs, int nDihs, float *scee, float *scnb, float *charge, float *ljA, float *ljB, int *atomType, int *nbparm, int nAtomTypes) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int t = threadIdx.x;
//	extern __shared__ float xyz_s[];
	int atom1;
	int atom2;
	int atom3;
	int atom4;
	int dihType;
	int k;
	float r1[nDim];
	float r2[nDim];
	float r3[nDim];
	float c11, c22, c33, c12, c13, c23;
	float t1, t2, t3, t4, t5, t6;
	float a, b;
	float f1, f4;
	float phi;
	float fdih;
	float hbox;
	float rMag, r6;
	int it, jt,  nlj;
	float f14e,f14v;
	
	//if (t < nAtoms*nDim) {
	//	xyz_s[t] = xyz[t];	
	//	__syncthreads();
	//}

	if (index < nDihs)
	{
		hbox = lbox/2.0;
		// determine two atoms to work  - these will be unique to each index
		atom1 = dihAtoms[index*5];
		atom2 = dihAtoms[index*5+1];
		atom3 = dihAtoms[index*5+2];
		atom4 = dihAtoms[index*5+3];
		dihType = dihAtoms[index*5+4];
		// Check to see if we want to compute the scaled 1-4 interaction
		if (atom3 > 0 && atom4 > 0) {
			//Scaled non-bonded interaction for 1-4
			rMag = 0.0f;
			for (k=0;k<nDim;k++) {
				r1[k] = xyz[atom1+k]-xyz[atom4+k];
				rMag += r1[k] * r1[k];
			}
			r6 = rMag*rMag*rMag;
			r6 = 1.0/r6;
			it = atomType[atom1];
			jt = atomType[atom4];
			nlj = nAtomTypes * (it-1) + jt - 1;
			nlj = nbparm[nlj];
			f14e = charge[atom1]*charge[atom4]/rMag/sqrtf(rMag)/scee[dihType];
			f14v = r6*(12.0f*ljA[nlj]*r6-6.0f*ljB[nlj])/scnb[dihType]/rMag;
			f14v = 0.0f;
			for (k=0;k<nDim;k++) {
				atomicAdd(&f[atom1+k], (f14e+f14v)*r1[k]);
				atomicAdd(&f[atom4+k], -(f14e+f14v)*r1[k]);
			}
		}
		if (atom4 < 0) { atom4 = -atom4;} // atom4 is negative if the torsion is improper
		if (atom3 < 0) { atom3 = -atom3;} // atom3 is negative if we don't want to compute the scaled 1-4 for this torsion

		c11 = 0.0f;
		c22 = 0.0f;
		c33 = 0.0f;
		c12 = 0.0f;
		c13 = 0.0f;
		c23 = 0.0f;
		for (k=0;k<nDim;k++) {
			r1[k] = xyz[atom1+k] - xyz[atom2+k];
			r2[k] = xyz[atom2+k] - xyz[atom3+k];
			r3[k] = xyz[atom3+k] - xyz[atom4+k];
			// assuming no more than one box away
			if (r1[k] > hbox) {
				r1[k] -= lbox;
			} else if (r1[k] < -hbox) {
				r1[k] += lbox;
			}
			if (r2[k] > hbox) {
				r2[k] -= lbox;
			} else if (r2[k] < -hbox) {
				r2[k] += lbox;
			}	
			if (r3[k] > hbox) {
				r3[k] -= lbox;
			} else if (r3[k] < -hbox) {
				r3[k] += lbox;
			}	
			c11 += r1[k]*r1[k];
			c22 += r2[k]*r2[k];
			c12 += r1[k]*r2[k];
			c33 += r3[k]*r3[k];
			c23 += r2[k]*r3[k];
			c13 += r1[k]*r3[k];
		}
		t1 = c13*c22-c12*c23;
		t2 = c11*c23-c12*c13;
		t3 = c12*c12-c11*c22;
		t4 = c22*c33-c23*c23;
		t5 = c13*c23-c12*c33;
		t6 = -t1;

		b = sqrtf(-t3*t4);
		a = t6/b;
		// make sure a is in the domain of the arccos
		if (a <= -1.0f) {
			fdih = 0.0;
		} else if (a >= 1.0f) {
			fdih = 0.0;	
		} else {
			phi = acos(a);
			fdih = dihNs[dihType] * dihKs[dihType] * sinf(dihNs[dihType]*phi-dihPs[dihType])/sinf(phi)*c22/b;
		}
		for (k=0;k<3;k++) {
			f1=fdih*(t1*r1[k]+t2*r2[k]+t3*r3[k])/t3;
			f4=-fdih*(t4*r1[k]+t5*r2[k]+t6*r3[k])/t4;
			atomicAdd(&f[atom1+k], f1);
			atomicAdd(&f[atom2+k], -(1.0f+c12/c22)*f1+c23/c22*f4);
			atomicAdd(&f[atom3+k], c12/c22*f1-(1.0f+c23/c22)*f4);
			atomicAdd(&f[atom4+k], f4);
		}

	}
}

/* C wrappers for kernels */

extern "C" void dih_force_cuda(float *xyz_d, float *f_d, int nAtoms, float lbox, int *dihAtoms_d, float *dihKs_d, float *dihNs_d, float *dihPs_d, int nDihs, float *scee_d, float *scnb_d, float *charge_d, float *ljA_d, float *ljB_d, int *atomType_d, int *nbparm_d, int nAtomTypes) {
	int blockSize;      // The launch configurator returned block size 
    	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    	int gridSize;       // The actual grid size needed, based on input size 

	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, dih_force_kernel, 0, nDihs); 

    	// Round up according to array size 
    	gridSize = (nDihs + blockSize - 1) / blockSize; 
	// run nondih cuda kernel
	//dih_force_kernel<<<gridSize, blockSize, nAtoms*nDim*sizeof(float)>>>(xyz_d, f_d, nAtoms, lbox, dihAtoms_d, dihKs_d, dihNs_d, dihPs_d, nDihs);
	dih_force_kernel<<<gridSize, blockSize>>>(xyz_d, f_d, nAtoms, lbox, dihAtoms_d, dihKs_d, dihNs_d, dihPs_d, nDihs, scee_d, scnb_d, charge_d, ljA_d, ljB_d, atomType_d, nbparm_d, nAtomTypes);

}

