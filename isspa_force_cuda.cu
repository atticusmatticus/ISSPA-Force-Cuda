#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "isspa_force_cuda.h"

#define nDim 3
#define MC 10
//Fast integer multiplication
#define MUL(a, b) __umul24(a, b)

//__global__ void init_rand(unsigned int long seed, hiprandState_t* states){
//	hiprand_init(seed,blockIdx.x,0,&states);
//}
// CUDA Kernels

__global__ void isspa_force_kernel(float *xyz, float *f, float *w, float *x0, float *g0, float *gr2, float *alpha, float *lj_A, float *lj_B, int *ityp, int nAtoms, int nMC, float lbox) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	float rnow;
	float prob;
	float attempt;
	float mc_pos[3];
	float mc_pos_atom[3];
	float x1, x2, r2;
	int atom;
	int atom2;
	int it;    // atom type of atom of interest
	int jt;    // atom type of other atom
	float gnow;
	float temp, dist2;
	int ev_flag, k;
	float rinv, r6, fs;
	float hbox;
	hiprandState_t state;

	if (index < nAtoms*nMC)
	{
		hbox = lbox/2.0;
		// get atom number of interest
		atom = index%nAtoms;
		it = ityp[atom];
		// initialize random number generator
		hiprand_init(0,blockIdx.x,index,&state);
		// select one point from 1D parabolic distribution
		rnow = 1.0f - 2.0f * hiprand_uniform(&state);
		prob = rnow*rnow;
		attempt = hiprand_uniform(&state);
		while (attempt < prob)
		{
			rnow = 1.0f - 2.0f * hiprand_uniform(&state);
			prob = rnow*rnow;
			attempt = hiprand_uniform(&state);
		}
		rnow = w[it] * rnow + x0[it];
		// select two points on surface of sphere
		x1 = 1.0f - 2.0f * hiprand_uniform(&state);
		x2 = 1.0f - 2.0f * hiprand_uniform(&state);
		r2 = x1*x1 + x2*x2;
		while (r2 > 1.0f) 
		{
			x1 = 1.0f - 2.0f * hiprand_uniform(&state);
                	x2 = 1.0f - 2.0f * hiprand_uniform(&state);
			r2 = x1*x1 + x2*x2;
		}
		// generate 3D MC pos based on position on surface of sphere and parabolic distribution in depth
		mc_pos[0] = rnow*(1.0f - 2.0f*r2);
		r2 = 2.0f * sqrtf(1.0f - r2);
		mc_pos[1] = rnow*x1*r2;
		mc_pos[2] = rnow*x2*r2;

		mc_pos_atom[0] = mc_pos[0] + xyz[atom*nDim];
		mc_pos_atom[1] = mc_pos[1] + xyz[atom*nDim+1];
		mc_pos_atom[2] = mc_pos[2] + xyz[atom*nDim+2];
		// compute density at MC point due to all other atoms
		gnow = 1.0f;
		ev_flag = 0;
		for (atom2=0;atom2<nAtoms;atom2++) 
		{
			if (atom2 != atom) 
			{
				jt = ityp[atom2];
				dist2 = 0.0f;
				for (k=0;k<nDim;k++) 
				{
					temp = mc_pos_atom[k] - xyz[atom2*nDim+k];
					if (temp > hbox) {
						temp -= (int)(temp/hbox) * lbox;
					} else if (temp < -hbox) {
						temp += (int)(temp/hbox) * lbox;
					}
					dist2 += temp*temp;
				}
				if (dist2 < gr2[jt*2]) {
					ev_flag = 1;	
					break;
				} else if (dist2 < gr2[jt*2+1]) {
					temp = sqrtf(dist2)-x0[jt];
					gnow *= (-alpha[jt] * temp*temp + g0[jt]);
				}
			}
		}
		
		if (ev_flag ==0) {
			rinv = 1.0f / rnow;
			r2 = rinv * rinv;
			r6 = r2 * r2 * r2;
			fs = gnow * r6 * (lj_B[it] - lj_A[it] * r6);
			atomicAdd(&f[atom*nDim], fs*mc_pos[0] / (float) nMC);
			atomicAdd(&f[atom*nDim+1], fs*mc_pos[1] / (float) nMC);
			atomicAdd(&f[atom*nDim+2], fs*mc_pos[2] / (float) nMC);
		}

	}
}

/* C wrappers for kernels */

extern "C" void isspa_force_cuda(float *xyz_d, float *f_d, float *w_d, float *x0_d, float *g0_d, float *gr2_d, float *alpha_d, float *lj_A_d, float *lj_B_d, int *ityp_d, int nAtoms, int nMC, float lbox) {
	int blockSize;      // The launch configurator returned block size 
    	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    	int gridSize;       // The actual grid size needed, based on input size 

	// zero force array on gpu
	hipMemset(f_d, 0.0,  nAtoms*nDim*sizeof(float));

	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, isspa_force_kernel, 0, nAtoms*nMC); 

    	// Round up according to array size 
    	gridSize = (nAtoms*nMC + blockSize - 1) / blockSize; 

	// run parabola random cuda kernal
	isspa_force_kernel<<<gridSize, blockSize>>>(xyz_d, f_d, w_d, x0_d, g0_d, gr2_d, alpha_d, lj_A_d, lj_B_d, ityp_d, nAtoms, nMC, lbox);

}

