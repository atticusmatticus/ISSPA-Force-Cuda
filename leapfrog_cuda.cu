#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "leapfrog_cuda.h"

#define nDim 3
//Fast integer multiplication
#define MUL(a, b) __umul24(a, b)

// CUDA Kernels

/*__device__ void thermo_kernel( float v*, float T, float mass, int atom, int block) {
	
	float rang;
	hiprandState_t state;
        hiprand_init(0,blockIdx.x,atom,&state);
	rang = hiprand_normal(&state);
	v[0] = rang*sqrtf(T/mass);
	rang = hiprand_normal(&state);
	v[1] = rang*sqrtf(T/mass);
	rang = hiprand_normal(&state);
	v[2] = rang*sqrtf(T/mass);

}*/

__global__ void leapfrog_kernel(float *xyz, float *v, float *f, float *mass, float T, float dt, float pnu, int nAtoms, float lbox, long long seed) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	float attempt;
	int k;
	hiprandState_t state;

	if (index < nAtoms)
	{
		// initialize random number generator
                hiprand_init(seed,index,0,&state);
		attempt = hiprand_uniform(&state);
		// anderson thermostat
		if (attempt < pnu) {
			//thermo_kernel(&v[index*nDim],T,mass[index],index, blockIdx);
			for (k=0;k<nDim;k++) {
				v[index*nDim+k] = hiprand_normal(&state) * sqrtf( T / mass[index] );
				v[index*nDim+k] += f[index*nDim+k]/mass[index]*dt/2.0;
				xyz[index*nDim+k] += v[index*nDim+k]*dt;
				if (xyz[index*nDim+k] > lbox) {
					xyz[index*nDim+k] -= (int) (xyz[index*nDim+k]/lbox) * lbox;
//					xyz[index*nDim+k] -= lbox;
				} else if (xyz[index*nDim+k] < 0.0f) {
					xyz[index*nDim+k] += (int) (-xyz[index*nDim+k]/lbox+1) * lbox;
//					xyz[index*nDim+k] += lbox;
				}
			}
		} else {
			for (k=0;k<nDim;k++) {
				v[index*nDim+k] += f[index*nDim+k]/mass[index]*dt;
				xyz[index*nDim+k] += v[index*nDim+k]*dt;
				if (xyz[index*nDim+k] > lbox) {
					xyz[index*nDim+k] -= (int) (xyz[index*nDim+k]/lbox) * lbox;
//					xyz[index*nDim+k] -= lbox;
				} else if (xyz[index*nDim+k] < 0.0f) {
					xyz[index*nDim+k] += (int) (-xyz[index*nDim+k]/lbox+1) * lbox;
//					xyz[index*nDim+k] += lbox;
				}
			}
		}
	}
}

/* C wrappers for kernels */

extern "C" void leapfrog_cuda(float *xyz_d, float *v_d, float *f_d, float *mass_d, float T, float dt, float pnu, int nAtoms, float lbox, long long seed) {
	int blockSize;      // The launch configurator returned block size 
    	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    	int gridSize;       // The actual grid size needed, based on input size 

	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, leapfrog_kernel, 0, nAtoms); 

    	// Round up according to array size 
    	gridSize = (nAtoms + blockSize - 1) / blockSize; 

	// run nonbond cuda kernel
	leapfrog_kernel<<<gridSize, blockSize>>>(xyz_d, v_d, f_d, mass_d, T, dt, pnu, nAtoms, lbox, seed);

}

