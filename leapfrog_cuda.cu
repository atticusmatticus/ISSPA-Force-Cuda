#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "leapfrog_cuda.h"
//#include "constants_cuda.cuh"
#include "constants.h"

//Fast integer multiplication
#define MUL(a, b) __umul24(a, b)

// CUDA Kernels

/*__device__ void thermo_kernel( float v*, float T, float mass, int atom, int block) {

	float rang;
	hiprandState_t state;
        hiprand_init(0,blockIdx.x,atom,&state);
	rang = hiprand_normal(&state);
	v[0] = rang*sqrtf(T/mass);
	rang = hiprand_normal(&state);
	v[1] = rang*sqrtf(T/mass);
	rang = hiprand_normal(&state);
	v[2] = rang*sqrtf(T/mass);

}*/

__global__ void leapfrog_kernel(float *xyz, float *v, float *f, float *mass, float T, float dt, float pnu, int nAtoms, float lbox, long long seed) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	float attempt;
	float force;
	float tempMass;
	float tempPos;
	float tempVel;
	int k;
	hiprandState_t state;

	if (index < nAtoms)
	{
		// initialize random number generator
  		hiprand_init(seed,index,0,&state);
		attempt = hiprand_uniform(&state);
		tempMass = __ldg(mass+index);
		// anderson thermostat
		if (attempt < pnu) {
			//thermo_kernel(&v[index*nDim],T,mass[index],index, blockIdx);
			for (k=0;k<nDim;k++) {
				force = __ldg(f+index*nDim+k);
				tempVel = hiprand_normal(&state) * sqrtf( T / tempMass );
				tempVel += force/tempMass*dt/2.0;
				v[index*nDim+k] = tempVel;
				//xyz[index*nDim+k] += temp*dt;
				tempPos = __ldg(xyz+index*nDim+k);
				tempPos += tempVel*dt;
				if (tempPos > lbox) {
					//xyz[index*nDik] -= (int) (xyz[index*nDim+k]/lbox) * lbox;
					tempPos -= lbox;
				} else if (tempPos < 0.0f) {
					//xyz[index*nDim+k] += (int) (-xyz[index*nDim+k]/lbox+1) * lbox;
					tempPos += lbox;
				}
				xyz[index*nDim+k] = tempPos;
			}
		} else {
			for (k=0;k<nDim;k++) {
				force = __ldg(f+index*nDim+k);
				tempVel = __ldg(v+index*nDim+k);
				tempVel += force/tempMass*dt;
				v[index*nDim+k] = tempVel;
				tempPos = __ldg(xyz+index*nDim+k);
			       	tempPos += tempVel*dt;
				if (tempPos > lbox) {
					tempPos -= lbox;
				} else if (tempPos < 0.0f) {
					tempPos += lbox;
				}
				xyz[index*nDim+k] = tempPos;
			}
		}
	}
}

/* C wrappers for kernels */

extern "C" void leapfrog_cuda(float *xyz_d, float *v_d, float *f_d, float *mass_d, float T, float dt, float pnu, int nAtoms, float lbox, long long seed) {
	int blockSize;      // The launch configurator returned block size
    	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    	int gridSize;       // The actual grid size needed, based on input size

	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, leapfrog_kernel, 0, nAtoms);

    	// Round up according to array size
    	gridSize = (nAtoms + blockSize - 1) / blockSize;
	// run nonbond cuda kernel
	leapfrog_kernel<<<gridSize, blockSize>>>(xyz_d, v_d, f_d, mass_d, T, dt, pnu, nAtoms, lbox, seed);

}
