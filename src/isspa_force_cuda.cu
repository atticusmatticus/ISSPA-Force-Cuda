#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "cuda_vector_routines.h"
#include "isspa_class.h"
#include "isspa_force_cuda.h"
#include "constants.h"

// constants
__constant__ int nTypes;
__constant__ int nMC;
__constant__ int nAtoms;
__constant__ int nPairs;
__constant__ float2 box;
__constant__ int nRs;
__constant__ float2 forceRparams;


// CUDA Kernels

__global__ void isspa_force_kernel(float4 *xyz, float4 *f, float4 *x0_w_vtot, float4 *gr2_g0_alpha, int *isspaTypes, float *forceTable, hiprandState *state) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int t = threadIdx.x;
	extern __shared__ float4 xyz_s[];
	int atom;
	float rnow;
        float prob;
       	float attempt;
	float x1, x2, r2;
	float4 mcpos;
	float4 mcr;
	float4 r;
	float dist2;
	int bin;
	int it, jt;
	int i;
	int atom2;
	float fs;
	float f1, f2, fracDist;
	float temp, gnow;
	float4 gr2_g0_alpha_l;
	float4 x0_w_vtot_l;
	hiprandState_t threadState;

	// copy atom positions to shared memory
	for (i=t;i<nAtoms;i+=blockDim.x) {
		xyz_s[i] = __ldg(xyz+i);
	}
	__syncthreads();
	// move on
	if (index < nAtoms*nMC)
	{
		// local variables
		// random number state - store in temporary variable
		threadState = state[index];
		// get atom number of interest
		atom = int(index/(float) nMC);
		mcpos = xyz_s[atom];
		// isspa type
		it = __ldg(isspaTypes+atom);
		// select one point from 1D parabolic distribution
		rnow = 1.0f - 2.0f * hiprand_uniform(&threadState);
		prob = rnow*rnow;
		attempt = hiprand_uniform(&threadState);
		while (attempt < prob)
		{
			rnow = 1.0f - 2.0f * hiprand_uniform(&threadState);
			prob = rnow*rnow;
			attempt = hiprand_uniform(&threadState);
		}
		x0_w_vtot_l = __ldg(x0_w_vtot+it);
		rnow = x0_w_vtot_l.y * rnow + x0_w_vtot_l.x;
		// select two points on surface of sphere
		x1 = 1.0f - 2.0f * hiprand_uniform(&threadState);
		x2 = 1.0f - 2.0f * hiprand_uniform(&threadState);
		r2 = x1*x1 + x2*x2;
		while (r2 > 1.0f)
		{
			x1 = 1.0f - 2.0f * hiprand_uniform(&threadState);
			x2 = 1.0f - 2.0f * hiprand_uniform(&threadState);
			r2 = x1*x1 + x2*x2;
		}
		// generate 3D MC pos based on position on surface of sphere and parabolic distribution in depth
		mcr.x = rnow*(1.0f - 2.0f*r2);
		r2 = 2.0f * sqrtf(1.0f - r2);
		mcr.y = rnow*x1*r2;
		mcr.z = rnow*x2*r2;
		mcpos += mcr;
		// initialize density to 1.0
		mcpos.w =  x0_w_vtot_l.z;
		// random state in global
		state[index] = threadState;

		// determine density at MC pos
		for (atom2=0;atom2<nAtoms;atom2++) {
			if (atom2 != atom) {	
				jt = __ldg(isspaTypes + atom2);
				gr2_g0_alpha_l = __ldg(gr2_g0_alpha+jt);
				r = min_image(mcpos - xyz_s[atom2],box.x,box.y);
				dist2 = r.x*r.x + r.y*r.y + r.z*r.z;
				if (dist2 < gr2_g0_alpha_l.x) {
					mcpos.w = 0.0f;
					break;
				} else if (dist2 < gr2_g0_alpha_l.y) {
					temp = sqrtf(dist2)-__ldg(x0_w_vtot+jt).x;
					gnow = (-gr2_g0_alpha_l.w * temp*temp + gr2_g0_alpha_l.z);
					mcpos.w *= gnow;
				}
			}
		}

		// add force to atoms
		if (mcpos.w > 0.0f) {
			bin = int ( (rnow-forceRparams.x)/forceRparams.y + 0.5f);
			// linearly interpolate between two force bins
			fracDist = (rnow - (forceRparams.x+bin*forceRparams.y)) / forceRparams.y;
			f1 = __ldg(forceTable+it*nRs+bin);
			f2 = __ldg(forceTable+it*nRs+bin+1);
			fs = f1*(1.0-fracDist)+f2*fracDist;
			fs *= mcpos.w;
			atomicAdd(&(f[atom].x), fs*mcr.x);
			atomicAdd(&(f[atom].y), fs*mcr.y);
			atomicAdd(&(f[atom].z), fs*mcr.z);
		}

	}
}


// CUDA Kernel wrapper
float isspa_force_cuda(float4 *xyz_d, float4 *f_d, isspa& isspas, int nAtoms_h) {

	float milliseconds;

	// timing
	hipEventRecord(isspas.isspaStart);
	

	// generate MC points
	isspa_force_kernel<<<isspas.mcGridSize, isspas.mcBlockSize,nAtoms_h*sizeof(float4)>>>(xyz_d, f_d, isspas.x0_w_vtot_d, isspas.gr2_g0_alpha_d, isspas.isspaTypes_d, isspas.isspaForceTable_d, isspas.randStates_d);


	// finish timing
	hipEventRecord(isspas.isspaStop);
	hipEventSynchronize(isspas.isspaStop);
	hipEventElapsedTime(&milliseconds, isspas.isspaStart, isspas.isspaStop);
	return milliseconds;

}

void isspa_grid_block(int nAtoms_h, int nPairs_h, float lbox_h, isspa& isspas) {

	int minGridSize;
	float2 box_h;

	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &isspas.mcBlockSize, isspa_force_kernel, 0, nAtoms_h*isspas.nMC);
    	// Round up according to array size
    	isspas.mcGridSize = (nAtoms_h*isspas.nMC + isspas.mcBlockSize - 1) / isspas.mcBlockSize;


	// set constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(nMC), &isspas.nMC, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nTypes), &isspas.nTypes, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nRs), &isspas.nRs, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nAtoms), &nAtoms_h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nPairs), &nPairs_h, sizeof(int));
	box_h.x = lbox_h;
	box_h.y = lbox_h/2.0;
	hipMemcpyToSymbol(HIP_SYMBOL(box), &box_h, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(forceRparams), &isspas.forceRparams, sizeof(float2));


}
