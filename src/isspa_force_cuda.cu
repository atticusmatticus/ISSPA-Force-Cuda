#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "cuda_vector_routines.h"
#include "isspa_class.h"
#include "isspa_force_cuda.h"
#include "constants.h"

using namespace std;

// constants
__constant__ int nTypes;
__constant__ int nMC;
__constant__ int nGRs;
__constant__ int nAtoms;
__constant__ int nPairs;
__constant__ float2 box;
__constant__ int nRs;
__constant__ float2 forceRparams;
__constant__ float2 gRparams;

// device functions

// CUDA Kernels

__global__ void isspa_force_kernel(float4 *xyz, float *vtot, float *rmax, int *isspaTypes, float *gTable, float *forceTable, float4 *f, hiprandState *state, float4 *out) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int t = threadIdx.x;
	extern __shared__ float4 xyz_s[];
	int atom;
	//float rnow;
        //float prob;
       	//float attempt;
	//float x1, x2, r2;
	//float temp;
	//float g1, g2;	//float gnow;
	float dist2, dist;
	float fs;
	float f1, f2, fracDist;
	float vtot_l;
	float rmax_l;
	float2 gRparams_l = gRparams;
	float4 mcpos;
	float4 r;
	float4 mcr;
	int bin;
	int it, jt;
	int i;
	int atom2;
	int igo;
	hiprandState_t threadState;

	
	// copy atom position to shared memory
	for (i=t;i<nAtoms;i+=blockDim.x) {
		xyz_s[i] = __ldg(xyz+i);
	}
	__syncthreads();
	// move on
	out[index].x = 2.0;

	if (index < nAtoms*nMC)
	{
	        // random number state - store in temporary variable
		threadState = state[index];
		// get atom number of interest
		atom = int(index/(float) nMC);
		mcpos = xyz_s[atom];
		// isspa type
		it = __ldg(isspaTypes+atom);
		vtot_l = __ldg(vtot+it);
		rmax_l = __ldg(rmax+it);
		igo = 0;
		// generate 3D MC pos based inside a sphere rnow
		mcr.x = rmax_l*(2.0f * hiprand_uniform(&threadState) - 1.0f);
		mcr.y = rmax_l*(2.0f * hiprand_uniform(&threadState) - 1.0f);
		mcr.z = rmax_l*(2.0f * hiprand_uniform(&threadState) - 1.0f);
		mcpos += mcr;
		// initialize density to 1.0
		mcpos.w = 1.0;
		// random state in global
		state[index] = threadState;

		
		// Get density from g table  
		// get atom number of interest
		for(atom2=0;atom2<nAtoms;atom2++){
		  if (atom2 != atom){
		    jt = __ldg(isspaTypes + atom2);
		    r = min_image(mcpos - xyz_s[atom2],box.x,box.y);
		    dist2 = r.x*r.x + r.y*r.y + r.z*r.z;
		    dist = sqrtf(dist2);
		    if (dist < rmax_l) {
		      // determine density bin of distance
		      bin = int ( (dist-gRparams_l.x)/gRparams_l.y );
		      // make sure bin is in limits of density table
		      if (bin >= (nGRs-1)) {
			continue;
		      }
		      else if (bin < 0) {
			mcpos.w *= 0.0;
			//break;
		      }
		      else {
			// linearly interpolate between two density bins
			//fracDist = (dist - (gRparams.x+bin*gRparams.y)) / gRparams.y;
			//g1 = __ldg(gTable+jt*nGRs+bin);
			//g2 = __ldg(gTable+jt*nGRs+bin+1);
			//mcpos.w *= g1*(1.0-fracDist)+g2*fracDist;
			mcpos.w *= __ldg(gTable + jt*nGRs+bin);
		      }
		      igo += 1;
		    }		      
		  }
		}
		// add force 
		if (mcpos.w > 0.0f) {
		  // get separation vector
		  // check dist is suppose to be rnow
		  bin = int ( (dist-forceRparams.x)/forceRparams.y + 0.5f);
		  // linearly interpolate between two force bins
		  fracDist = (dist - (forceRparams.x+bin*forceRparams.y)) / forceRparams.y;
		  f1 = __ldg(forceTable+it*nRs+bin);
		  f2 = __ldg(forceTable+it*nRs+bin+1);
		  fs = f1*(1.0-fracDist)+f2*fracDist;
		  fs *= mcpos.w * vtot_l / igo; 
		  atomicAdd(&(f[atom].x), fs*mcr.x);
		  atomicAdd(&(f[atom].y), fs*mcr.y);
		  atomicAdd(&(f[atom].z), fs*mcr.z);
		}		
	}
}



/* C wrappers for kernels */

float isspa_force_cuda(float4 *xyz_d, float4 *f_d, isspa& isspas, int nAtoms_h) {

	float milliseconds;

	float4 out_h[nAtoms_h*isspas.nMC];

	
	// timing
	hipEventRecord(isspas.isspaStart);

	// compute isspa force
	isspa_force_kernel<<<isspas.mcGridSize, isspas.mcBlockSize, nAtoms_h*sizeof(float4)>>>(xyz_d, isspas.vtot_d, isspas.rmax_d, isspas.isspaTypes_d, isspas.isspaGTable_d, isspas.isspaForceTable_d, f_d, isspas.randStates_d, isspas.out_d);
	// DEBUG
	hipMemcpy(out_h, isspas.out_d, nAtoms_h*isspas.nMC*sizeof(float4), hipMemcpyDeviceToHost);
	//for (int i=0;i<nAtoms_h*isspas.nMC; i++)
	for (int i=0;i<2*isspas.nMC; i++)
	{
	  printf("C %10.6f %10.6f %10.6f %10.6f\n", out_h[i].x, out_h[i].y, out_h[i].z, out_h[i].w);
	}

	// finish timing
	hipEventRecord(isspas.isspaStop);
	hipEventSynchronize(isspas.isspaStop);
	hipEventElapsedTime(&milliseconds, isspas.isspaStart, isspas.isspaStop);
	return milliseconds;

}

void isspa_grid_block(int nAtoms_h, int nPairs_h, float lbox_h, isspa& isspas) {

	int minGridSize;
	float2 box_h;
	
	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &isspas.mcBlockSize, isspa_force_kernel, 0, nAtoms_h*isspas.nMC);
    	// Round up according to array size
    	isspas.mcGridSize = (nAtoms_h*isspas.nMC + isspas.mcBlockSize - 1) / isspas.mcBlockSize;

	// fill box with box and half box length
	box_h.x = lbox_h;
	box_h.y = lbox_h/2.0;
	
	// set constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(nMC), &isspas.nMC, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nTypes), &isspas.nTypes, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nRs), &isspas.nRs, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nGRs), &isspas.nGRs, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nAtoms), &nAtoms_h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nPairs), &nPairs_h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(box), &box_h, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(forceRparams), &isspas.forceRparams, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(gRparams), &isspas.gRparams, sizeof(float2));

}
