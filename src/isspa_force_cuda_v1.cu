#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "cuda_vector_routines.h"
#include "isspa_class.h"
#include "isspa_force_cuda.h"
#include "constants.h"

// constants
__constant__ int nTypes;
__constant__ int nMC;
__constant__ int nAtoms;
__constant__ int nPairs;
__constant__ float2 box;
__constant__ int nRs;
__constant__ float2 forceRparams;

// device functions

__device__ float atomicMul(float* address, float val) 
{ 
	unsigned int* address_as_u = (unsigned int*)address; 
	unsigned int old = *address_as_u, assumed; 
	do { 
		assumed = old; 
		old = atomicCAS(address_as_u, assumed, __float_as_uint(val * __uint_as_float(assumed))); 
	} while (assumed != old); return __uint_as_float(old);
}


// CUDA Kernels

__global__ void isspa_force_kernel(float4 *xyz, float2 *x0_w, int *isspaTypes, float4 *gr2_g0_alpha, float *forceTable, float4 *f, float *vtot, hiprandState *state) {
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int t = threadIdx.x;
	extern __shared__ float2 params_s[];
	int atom;
	float rnow;
        float prob;
       	float attempt;
	float x1, x2, r2;
	float dist,dist2;
	float fs;
	float f1, f2, fracDist;
	float temp;
	float gnow;
	float4 gr2_g0_alpha_l;
	float4 mcpos;
	float4 r;
	int bin;
	int it, jt;
	int i;
	int atom2;
	hiprandState_t threadState;

	
	// copy density parameters to shared memory
	for (i=t;i<nTypes;i+=blockDim.x) {
		params_s[i] = __ldg(x0_w+i);
	}
	__syncthreads();
	// move on
	if (index < nAtoms*nMC)
	{

		// random number state - store in temporary variable
		threadState = state[index];
		// get atom number of interest
		atom = int(index/(float) nMC);
		mcpos = __ldg(xyz+atom);
		// isspa type
		it = __ldg(isspaTypes+atom);
		// select one point from 1D parabolic distribution
		rnow = 1.0f - 2.0f * hiprand_uniform(&threadState);
		prob = rnow*rnow;
		attempt = hiprand_uniform(&threadState);
		while (attempt < prob)
		{
			rnow = 1.0f - 2.0f * hiprand_uniform(&threadState);
			prob = rnow*rnow;
			attempt = hiprand_uniform(&threadState);
		}
		rnow = params_s[it].y * rnow + params_s[it].x;
		// select two points on surface of sphere
		x1 = 1.0f - 2.0f * hiprand_uniform(&threadState);
		x2 = 1.0f - 2.0f * hiprand_uniform(&threadState);
		r2 = x1*x1 + x2*x2;
		while (r2 > 1.0f)
		{
			x1 = 1.0f - 2.0f * hiprand_uniform(&threadState);
			x2 = 1.0f - 2.0f * hiprand_uniform(&threadState);
			r2 = x1*x1 + x2*x2;
		}
		// generate 3D MC pos based on position on surface of sphere and parabolic distribution in depth
		mcpos.x = mcpos.x + rnow*(1.0f - 2.0f*r2);
		r2 = 2.0f * sqrtf(1.0f - r2);
		mcpos.y = mcpos.y + rnow*x1*r2;
		mcpos.z = mcpos.z + rnow*x2*r2;
		// initialize density to 1.0
		mcpos.w = 1.0;

		// random state in global
		state[index] = threadState;

		// get density
		// get atom number of interest
		for(atom2=0;atom2<nAtoms;atom2++){
		  if (atom2 != atom){
		    jt = __ldg(isspaTypes + atom2);
		    gr2_g0_alpha_l = __ldg(gr2_g0_alpha+jt);
		    r = min_image(mcpos - __ldg(xyz+atom2),box.x,box.y);
		    dist2 = r.x*r.x + r.y*r.y + r.z*r.z;
		    // if inside of excluded volume set to zero
		    if (dist2 < gr2_g0_alpha_l.x) {
		      mcpos.w = 0.0f;
		    } else if (dist2 < gr2_g0_alpha_l.y) {
		      temp = sqrtf(dist2)-params_s[jt].x;
		      gnow = (-gr2_g0_alpha_l.w * temp*temp + gr2_g0_alpha_l.z);
		      mcpos.w *=gnow;
		    }
		  }
		}
		// add force 
		if (mcpos.w > 0.0f) {
		  // get separation vector
		  r = mcpos - __ldg(xyz+atom);
		  r2 = r.x*r.x + r.y*r.y + r.z*r.z;
		  dist = sqrtf(r2);
		  bin = int ( (dist-forceRparams.x)/forceRparams.y + 0.5f);
		  // linearly interpolate between two force bins
		  fracDist = (dist - (forceRparams.x+bin*forceRparams.y)) / forceRparams.y;
		  f1 = __ldg(forceTable+it*nRs+bin);
		  f2 = __ldg(forceTable+it*nRs+bin+1);
		  fs = f1*(1.0-fracDist)+f2*fracDist;
		  fs *= mcpos.w * vtot[it];
		  atomicAdd(&(f[atom].x), fs*r.x);
		  atomicAdd(&(f[atom].y), fs*r.y);
		  atomicAdd(&(f[atom].z), fs*r.z);
		}
		

	}

}



/* C wrappers for kernels */

float isspa_force_cuda(float4 *xyz_d, float4 *f_d, isspa& isspas) {

	float milliseconds;

	// timing
	hipEventRecord(isspas.isspaStart);
	

	// compute isspa force
	isspa_force_kernel<<<isspas.mcGridSize, isspas.mcBlockSize,isspas.nTypes*sizeof(float2)>>>(xyz_d,isspas.x0_w_d, isspas.isspaTypes_d, isspas.gr2_g0_alpha_d, isspas.isspaForceTable_d, f_d, isspas.vtot_d, isspas.randStates_d);

	  // generate MC pointsz
	//isspa_mc_kernel<<<isspas.mcGridSize, isspas.mcBlockSize,isspas.nTypes*sizeof(float2)>>>(xyz_d, isspas.mcpos_d, isspas.x0_w_d, isspas.isspaTypes_d, isspas.randStates_d);
	//printf("MC points generated\n");
	// compute density at each mc point
	//isspa_density_kernel<<<isspas.gGridSize, isspas.gBlockSize,isspas.nTypes*sizeof(float4)>>>(xyz_d, isspas.mcpos_d, isspas.x0_d, isspas.gr2_g0_alpha_d, isspas.isspaTypes_d);
	// add to forces
	//isspa_force_kernel<<<isspas.mcGridSize, isspas.mcBlockSize,isspas.nTypes*sizeof(float)>>>(xyz_d, f_d, isspas.mcpos_d, isspas.vtot_d, isspas.isspaTypes_d, isspas.isspaForceTable_d);


	// finish timing
	hipEventRecord(isspas.isspaStop);
	hipEventSynchronize(isspas.isspaStop);
	hipEventElapsedTime(&milliseconds, isspas.isspaStart, isspas.isspaStop);
	return milliseconds;

}

void isspa_grid_block(int nAtoms_h, int nPairs_h, float lbox_h, isspa& isspas) {

	int minGridSize;
	float2 box_h;
	
	// determine gridSize and blockSize
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &isspas.mcBlockSize, isspa_force_kernel, 0, nAtoms_h*isspas.nMC);
    	// Round up according to array size
    	isspas.mcGridSize = (nAtoms_h*isspas.nMC + isspas.mcBlockSize - 1) / isspas.mcBlockSize;

	// fill box with box and half box length
	box_h.x = lbox_h;
	box_h.y = lbox_h/2.0;
	
	// set constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(nMC), &isspas.nMC, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nTypes), &isspas.nTypes, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nRs), &isspas.nRs, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nAtoms), &nAtoms_h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nPairs), &nPairs_h, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(box), &box_h, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(forceRparams), &isspas.forceRparams, sizeof(float2));


}
